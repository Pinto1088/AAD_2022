//Implement a Parallel ODD-Even Sort algorithm using GPU or ARM equivalent.
 
#include <hip/hip_runtime.h>
#include"iostream"  
 using namespace std;  
  __global__ void sort(int *in, int n)  
  {  
       int temp;  
       bool oddeven=true;  
       __shared__ bool oddswap, evenswap;          // shared variables only, don't use private ones  
       while(true)  
       {  
            if(oddeven==true)                         // Even turn  
            {  
                 __syncthreads();  
                 evenswap=false;  
                 __syncthreads();  
                      int idx=threadIdx.x;  
                      if(idx<(n/2))  
                      {  
                           if (in[2*idx]>in[2*idx+1] )          // swapping  
                           {  
                                temp = in[2*idx];  
                                in[2*idx]=in[2*idx+1];              
                                in[2*idx+1]=temp;  
                                evenswap=true;  

                           }  
                      }  
                 __syncthreads();  
            }  
            else                                        // odd turn  
            {  
                 __syncthreads();  
                 oddswap=false;  
                __syncthreads();  
                      int idx=threadIdx.x;  
                     if(idx<(n/2))  
                      {  
                           if (in[2*idx+1]>in[2*idx+2] && (2*idx+2)<n)     // swapping  
                           {  
                                temp= in[2*idx+1];  
                                in[2*idx+1]=in[2*idx+2];  
                                in[2*idx+2]=temp;  
                                oddswap=true;  
                           }  
                      }  
                 __syncthreads();  
            }  
            if(!(oddswap||evenswap))               // when to stop  
                break;  
            oddeven=!oddeven;  
       }  
  }  
  int main()  
  {  
       int *a, *d_a, n;  
      cout<<"\nEnter no. of elements you want to sort: ";  
       cin>>n;  
       size_t size = sizeof(int)*n;  
       a = new int[n];                         // allocate size for a on main memory(RAM)  
       hipMalloc((void**)&d_a, size);             // allocate size for a on global memory(GPU memory)  
       cout<<"\n\nEnter no.s to be sorted: \n";  
       for(int i=0;i<n;i++)  
       {  
            cin>>a[i];  
       }  
       hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);  
      sort<<<1,n/2>>>(d_a,n);  
       hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);  
       cout<<"\nSorted array is: \n";  
       for (int i=0;i<n;i++)  
      {  
           cout<<a[i]<<" ";  
      }  
      delete(a);               // free allocated space from main memory & global memory  
      hipFree(d_a);  
       return 0;  
 } 
/*
 Output:
 shri@shri-System-Product-Name:~$ nvcc ODDEVEN1.cu
shri@shri-System-Product-Name:~$ ./a.out

Enter no. of elements you want to sort: 10


Enter no.s to be sorted: 
10
9
8
7
6
5
4
3
2
1

Sorted array is: 
1 2 3 4 5 6 7 8 9 10 
shri@shri-System-Product-Name:~$ 
*/
